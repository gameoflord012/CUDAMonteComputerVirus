#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <random>
#include <stdlib.h>
#include <time.h>

//for __syncthreads()
#include <hip/device_functions.h>

#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;

const size_t EXPECTED_TEST_COUNT = 100;
const unsigned int INITIAL_INFECTED_COUNT = 1;
const unsigned int THREADS_PER_BLOCK = 128;
const unsigned int BLOCKS_COUNT_PER_numSMs = 32;

__device__ float d_probs[20];

__global__
void monte_simp(hiprandState_t* states, size_t* result, size_t n)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    for (size_t i = threadId; i < n; i += gridDim.x * blockDim.x)
    {
        int infectedCount = INITIAL_INFECTED_COUNT;

        do
        {
            atomicAdd(result, (size_t)1);

            int newCasesCount = 0;
            for (int uninfectedCount = 20 - infectedCount; uninfectedCount--;)
            {
                float prob = hiprand_uniform(&states[threadId]);
                newCasesCount += prob < d_probs[infectedCount];
            }

            infectedCount += newCasesCount;
        } while ((infectedCount -= 5) > 0);
    }
}

__global__
void curand_init_kernel(hiprandState_t* states, int seed)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, i, 0, &states[i]);
}

int main()
{
    // Init kernel
    srand(time(NULL));

    int numSMs, device;
    CUDA_CALL(hipGetDevice(&device));
    CUDA_CALL(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, device));

    int THREADS_COUNT = numSMs * BLOCKS_COUNT_PER_numSMs * THREADS_PER_BLOCK;
    hiprandState_t* d_states;
    CUDA_CALL(hipMalloc((void**)&d_states, THREADS_COUNT * sizeof(hiprandState)));

    curand_init_kernel << <numSMs * BLOCKS_COUNT_PER_numSMs, THREADS_PER_BLOCK >> > (d_states, rand());

    // Initialize d_A
    size_t* d_A;
    CUDA_CALL(hipMalloc((void**)&d_A, sizeof(size_t)));
    CUDA_CALL(hipMemset(d_A, 0, sizeof(size_t)));

    // Calculate probs
    float h_probs[20];
    float currentProb = 1.;
    for (int i = 0; i < 20; i++)
    {
        h_probs[i] = 1. - currentProb;
        currentProb *= .9;
    }

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_probs), h_probs, 20 * sizeof(float)));

    // Run simulation kernel
    monte_simp<<<numSMs * BLOCKS_COUNT_PER_numSMs, THREADS_PER_BLOCK>>>(d_states, d_A, EXPECTED_TEST_COUNT);

    // print result from device memory
    size_t h_A;
    CUDA_CALL(hipMemcpy(&h_A, d_A, sizeof(size_t), hipMemcpyDeviceToHost));

    printf("Threads count: %d\n", THREADS_COUNT);
    printf("%llu %llu\n", h_A, EXPECTED_TEST_COUNT);
    printf("%.2f\n", (float)(h_A) / EXPECTED_TEST_COUNT);

    // Free memory
    hipFree(d_states);
    hipFree(d_A);

    // Check for any errors
    cudaCheckErrors("Don't know wtf");
}