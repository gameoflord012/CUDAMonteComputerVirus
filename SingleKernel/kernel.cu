#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <random>
#include <stdlib.h>
#include <time.h>
#include <sstream>
#include <chrono>

//for __syncthreads()
#include <hip/device_functions.h>

#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;

const size_t DEFAULT_TESTS_COUNT = 100;
const unsigned int DEFAULT_THREADS_COUNT_PER_BLOCK = 128;

const unsigned int INITIAL_INFECTED_COUNT = 1;

__device__ float d_probs[20];

__global__
void monte_simp(size_t* result, size_t n)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t s;
    hiprand_init(clock64(), threadId, 0 ,&s);

    for (size_t i = threadId; i < n; i += gridDim.x * blockDim.x)
    {
        int infectedCount = INITIAL_INFECTED_COUNT;

        do
        {
            atomicAdd(result, (size_t)1);

            int newCasesCount = 0;
            for (int uninfectedCount = 20 - infectedCount; uninfectedCount--;)
            {
                float prob = hiprand_uniform(&s);
                newCasesCount += prob < d_probs[infectedCount];
            }

            infectedCount += newCasesCount;
        } while ((infectedCount -= 5) > 0);
    }
}

size_t get_unsigned_num(string msg, size_t defaultValue)
{
    cout << msg;
    string input;
    getline(cin, input);
    stringstream ss(input);
    size_t result;
    if (ss >> result)
        return result;
    return defaultValue;
}

int main()
{
    // Get numSMs and device
    int numSMs, device;
    CUDA_CALL(hipGetDevice(&device));
    CUDA_CALL(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, device));
    
    // Get inputs
    const size_t tests_count = get_unsigned_num("tests_count: ", DEFAULT_TESTS_COUNT);
    const size_t threads_count_per_block = get_unsigned_num("threads_per_block: ", DEFAULT_THREADS_COUNT_PER_BLOCK);

    const unsigned int default_blocks_count = (tests_count + threads_count_per_block - 1) / threads_count_per_block;
    const size_t blocks_count_per_numSMs = get_unsigned_num("blocks_count_per_numSMs: ", (default_blocks_count + numSMs - 1) / numSMs);

    const unsigned int threads_count = numSMs * blocks_count_per_numSMs * threads_count_per_block;

    // Print datas
    cout << endl;
    cout << "Datas:" << endl;
    cout << "   tests_count = " << tests_count << endl;
    cout << "   blocks_count_per_numSMs = " << blocks_count_per_numSMs << endl;
    cout << "   numSMs = " << numSMs << endl;
    cout << "   threads_count_per_block = " << threads_count_per_block << endl;
    cout << "   threads_count = " << threads_count << endl;
    cout << endl;

    // Initialize d_A
    size_t* d_A;
    CUDA_CALL(hipMalloc((void**)&d_A, sizeof(size_t)));
    CUDA_CALL(hipMemset(d_A, 0, sizeof(size_t)));

    // Calculate probs
    float h_probs[20];
    float currentProb = 1.;
    for (int i = 0; i < 20; i++)
    {
        h_probs[i] = 1. - currentProb;
        currentProb *= .9;
    }

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_probs), h_probs, 20 * sizeof(float)));

    // Run simulation kernel
    cout << "Running kernel \"monte_simp()\"..." << endl;
    clock_t tStart = clock();
    monte_simp << <numSMs * blocks_count_per_numSMs, threads_count_per_block >> > (d_A, tests_count);
    CUDA_CALL(hipDeviceSynchronize());
    printf("Time taken: %.2fs\n\n", (double)(clock() - tStart) / CLOCKS_PER_SEC);

    
    // print result from device memory
    size_t h_A;
    CUDA_CALL(hipMemcpy(&h_A, d_A, sizeof(size_t), hipMemcpyDeviceToHost));

    cout << "Results: " << endl;
    printf("    A) %llu / %llu = %.2f\n", h_A, tests_count, (float)(h_A) / tests_count);

    // Free memory
    hipFree(d_A);

    // Check for any errors
    cudaCheckErrors("Don't know wtf");
}