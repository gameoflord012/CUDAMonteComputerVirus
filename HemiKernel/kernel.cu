#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <random>
#include <stdlib.h>
#include <time.h>
#include <sstream>
#include <chrono>

#include "hemi/hemi.h"
#include "hemi/launch.h"
#include "hemi/grid_stride_range.h"

//for __syncthreads()
#include <hip/device_functions.h>

#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define DEBUG

using namespace std;

const size_t DEFAULT_TESTS_COUNT = 10;

const unsigned int INITIAL_INFECTED_COUNT = 1;

__device__ float d_probs[20];

HEMI_LAUNCHABLE
void monte_simp(size_t* result, size_t n)
{
    hiprandState_t s;
    hiprand_init(clock64(), hemi::globalThreadIndex(), 0, &s);

    for (auto i : hemi::grid_stride_range<size_t>(0, n))
    {
        int infectedCount = INITIAL_INFECTED_COUNT;

        do
        {
            atomicAdd(result, (size_t)1);

            int newCasesCount = 0;
            for (int uninfectedCount = 20 - infectedCount; uninfectedCount--;)
            {
                float prob = hiprand_uniform(&s);
                newCasesCount += prob < d_probs[infectedCount];
            }

            infectedCount += newCasesCount;
        } while ((infectedCount -= 5) > 0);
    }
}

size_t get_unsigned_num(string msg, size_t defaultValue)
{
    cout << msg;
    string input;
    getline(cin, input);
    stringstream ss(input);
    size_t result;
    if (ss >> result)
        return result;
    return defaultValue;
}

int main()
{
    // Get inputs
    const size_t tests_count = get_unsigned_num("tests_count: ", DEFAULT_TESTS_COUNT);

    // Print datas
    cout << endl;
    cout << "Datas:" << endl;
    cout << "   tests_count = " << tests_count << endl;
    cout << endl;

    // Initialize d_A
    size_t* d_A;
    checkCuda(hipMalloc((void**)&d_A, sizeof(size_t)));
    checkCuda(hipMemset(d_A, 0, sizeof(size_t)));

    // Calculate probs
    float h_probs[20];
    float currentProb = 1.;
    for (int i = 0; i < 20; i++)
    {
        h_probs[i] = 1. - currentProb;
        currentProb *= .9;
    }

    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(d_probs), h_probs, 20 * sizeof(float)));

    // Run simulation kernel
    cout << "Running kernel \"monte_simp()\"..." << endl;
    clock_t tStart = clock();
    hemi::hipLaunchByPtr(monte_simp, d_A, tests_count);
    checkCuda(hipDeviceSynchronize());
    printf("Time taken: %.2fs\n\n", (double)(clock() - tStart) / CLOCKS_PER_SEC);


    // print result from device memory
    size_t h_A;
    checkCuda(hipMemcpy(&h_A, d_A, sizeof(size_t), hipMemcpyDeviceToHost));

    cout << "Results: " << endl;
    printf("    A) %llu / %llu = %.2f\n", h_A, tests_count, (float)(h_A) / tests_count);

    // Free memory
    checkCuda(hipFree(d_A));

    // Check for any errors
    checkCudaErrors();
}