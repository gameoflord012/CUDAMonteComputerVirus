#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <random>
#include <stdlib.h>
#include <time.h>

//for __syncthreads()
#include <hip/device_functions.h>

#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

using namespace std;

const size_t EXPECTED_TEST_COUNT = 100;
const unsigned int THREADS_PER_BLOCK = 128;
const unsigned int BLOCKS_COUNT_PER_numSMs = 32;

const unsigned int INITIAL_INFECTED_COUNT = 1;

__device__ float d_probs[20];

__global__
void uniform_dis(hiprandState_t* states, unsigned int* result, float p)
{
    float prob = hiprand_uniform(states + threadIdx.x);
    atomicAdd(result, prob < p);
}

__global__
void monte_simp(hiprandState_t* states, unsigned int* counters, size_t* result, size_t n)
{
    unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    for (size_t i = threadId; i < n; i += gridDim.x * blockDim.x)
    {
        int infectedCount = INITIAL_INFECTED_COUNT;

        do
        {
            atomicAdd(result, (size_t)1);

            counters[threadId] = 0;
            uniform_dis << <1, 20 >> > (states + threadId * 20, counters + threadId, d_probs[infectedCount]);
            infectedCount += counters[threadId];
        } while ((infectedCount -= 5) > 0);
    }
}

__global__
void init_kernel_states(hiprandState_t* states, int seed)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, i, 0, states + i);
}

int main()
{
    // Init kernels
    srand(time(NULL));

    int numSMs, device;
    CUDA_CALL(hipGetDevice(&device));
    CUDA_CALL(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, device));

    unsigned int THREADS_COUNT = numSMs * BLOCKS_COUNT_PER_numSMs * THREADS_PER_BLOCK;
    hiprandState_t* d_states;
    CUDA_CALL(hipMalloc((void**)&d_states, 20 * THREADS_COUNT * sizeof(hiprandState)));

    init_kernel_states << <numSMs * BLOCKS_COUNT_PER_numSMs * 20, THREADS_PER_BLOCK >> > (d_states, rand());

    unsigned int* counters;
    CUDA_CALL(hipMalloc((void**)&counters, THREADS_COUNT * sizeof(unsigned int)));

    // Initialize d_A
    size_t* d_A;
    CUDA_CALL(hipMalloc((void**)&d_A, sizeof(size_t)));
    CUDA_CALL(hipMemset(d_A, 0, sizeof(size_t)));

    // Calculate probs
    float h_probs[20];
    float currentProb = 1.;
    for (int i = 0; i < 20; i++)
    {
        h_probs[i] = 1. - currentProb;
        currentProb *= .9;
    }

    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_probs), h_probs, 20 * sizeof(float)));

    // Run simulation kernel
    monte_simp << <numSMs * BLOCKS_COUNT_PER_numSMs, THREADS_PER_BLOCK >> > (d_states, counters, d_A, EXPECTED_TEST_COUNT);

    // print result from device memory
    size_t h_A;
    CUDA_CALL(hipMemcpy(&h_A, d_A, sizeof(size_t), hipMemcpyDeviceToHost));

    printf("Threads count: %d\n", THREADS_COUNT);
    printf("%llu %llu\n", h_A, EXPECTED_TEST_COUNT);
    printf("%.2f\n", (float)(h_A) / EXPECTED_TEST_COUNT);

    // Free memory
    hipFree(d_states);
    hipFree(d_A);

    // Check for any errors
    cudaCheckErrors("Don't know wtf");
}